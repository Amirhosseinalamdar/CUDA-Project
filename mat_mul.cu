#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <math.h>
#include <stdlib.h>

#define MAX_FILE_NAME 64

/**
 * @brief scan the matrix from file
 *
 * @param n1
 * @param n2
 * @param matrix
 * @param file
 */
void get_matrix(int n1, int n2, float *matrix, FILE *file)
{
    for (int i = 0; i < n1; i++)
    {
        for (int j = 0; j < n2; j++)
        {
            fscanf(file, "%f ", matrix + i * n1 + j);
        }
    }
}

/**
 * @brief Get the inputs from given file
 *
 * @param M1
 * @param M2
 * @param n1
 * @param n2
 * @param n3
 */
void get_inputs(float **M1, float **M2, int *n1, int *n2, int *n3)
{
    char file_path[MAX_FILE_NAME];
    scanf("%s", file_path);
    FILE *file = fopen(file_path, "r");
    fscanf(file, "%d %d %d\n", n1, n2, n3);

    *M1 = (float *)malloc((*n1) * (*n2) * sizeof(float));
    *M2 = (float *)malloc((*n2) * (*n3) * sizeof(float));

    get_matrix(*n1, *n2, *M1, file);
    get_matrix(*n2, *n3, *M2, file);
}

/**
 * @brief compare computed results with the real results (in low dimensions)
 *
 * @param Ans
 * @param A
 * @param B
 * @param n1
 * @param n2
 * @param n3
 */
void verify(float *Ans, float *A, float *B, int n1, int n2, int n3)
{
    for (int i = 0; i < n1; i++)
    {
        for (int j = 0; j < n3; j++)
        {
            float sum = 0;
            for (int k = 0; k < n2; k++)
            {
                sum += A[k * n1 + i] * B[j * n2 + k];
            }
            printf("%.2f ", sum);
        }
        printf("\n");
    }
    for (int i = 0; i < n1; i++)
    {
        for (int j = 0; j < n3; j++)
        {
            printf("%.2f ", Ans[j * n1 + i]);
        }
        printf("\n");
    }
}

int main()
{
    int m, n, k;
    float *A, *B, *C;

    get_inputs(&A, &B, &m, &n, &k);

    float *A_d, *B_d, *C_d;
    C = (float *)malloc(m * k * sizeof(float));
    hipMalloc(&A_d, m * n * sizeof(float));
    hipMalloc(&B_d, n * k * sizeof(float));
    hipMalloc(&C_d, m * k * sizeof(float));

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipMemcpy(A_d, A, m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, n * k * sizeof(float), hipMemcpyHostToDevice);

    float alpha = 1.0f;
    float beta = 0.0f;

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, k, n, &alpha, A_d, m, B_d, n, &beta, C_d, m);

    hipMemcpy(C, C_d, m * k * sizeof(float), hipMemcpyDeviceToHost);

    // verify(C, A, B, m, n, k);

    free(A);
    free(B);
    free(C);
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

    return 0;
}