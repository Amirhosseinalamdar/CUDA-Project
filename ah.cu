#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <math.h>
#include <stdlib.h>

#define MAX_FILE_NAME 64

/**
 * @brief scan the matrix from file
 *
 * @param n1
 * @param n2
 * @param matrix
 * @param file
 */
void get_matrix(int n1, int n2, float *matrix, FILE *file)
{
    for (int i = 0; i < n1; i++)
    {
        for (int j = 0; j < n2; j++)
        {
            fscanf(file, "%f ", matrix + i * n2 + j);
        }
    }
}

/**
 * @brief Get the inputs from given file
 *
 * @param M1
 * @param M2
 * @param n1
 * @param n2
 * @param n3
 */
void get_inputs(float **M1, float **M2, int *n1, int *n2, int *n3)
{
    char file_path[MAX_FILE_NAME];
    // printf("input path to txt file:\n");
    // scanf("%s", file_path);
    FILE *file = fopen("sample.txt", "r");
    fscanf(file, "%d %d %d\n", n1, n2, n3);

    *M1 = (float *)malloc((*n1) * (*n2) * sizeof(float));
    *M2 = (float *)malloc((*n2) * (*n3) * sizeof(float));

    get_matrix(*n1, *n2, *M1, file);
    get_matrix(*n2, *n3, *M2, file);
    fclose(file);
}

/**
 * @brief compare computed results with the real results (in low dimensions)
 *
 * @param Ans
 * @param A
 * @param B
 * @param n1
 * @param n2
 * @param n3
 */
int verify(float *Ans, float *A, float *B, int n1, int n2, int n3)
{
    // int a[12];
    // for (int i = 0; i < n1; i++)
    // {
    //     for (int j = 0; j < n3; j++)
    //     {
    //         float sum = 0;
    //         for (int k = 0; k < n2; k++)
    //         {
    //             sum += A[i * n2 + k] * B[k * n3 + j];
    //         }
    //         a[i*n3 + j] = sum;
    //         printf("%.2f ", sum);
    //     }
    // }
    // for(int i=0 ;i< 12;i++){
    //     printf("%.2f ", Ans[i]);
    // }
    if ((Ans[0] - (-90.13) < 0.01) && (Ans[1] - 54.24 < 0.01) && (Ans[4] - (-34.77) < 0.01)) 
    {
        printf("%.2f %.2f %.2f", Ans[0], Ans[1], Ans[4]);
        return 1;
    }
    else
        return 0;
}
bool inc(int s[])
{
    int i = 0;
    s[i] += 1;
    while (s[i] == 3 && i < 5)
    {
        s[i] = 0;
        i++;
        s[i] += 1;
    }

    if (s[5] == 3)
        return 0;
    else
        return 1;
}
int set_val(int m, int n, int k, int num)
{
    if (num % 3 == 0)
        return m;
    else if (num % 3 == 1)
        return n;
    else if (num % 3 == 2)
        return k;

    printf("ridii");
    return 0;
}

int main()
{
    int state[6] = {0, 0, 0, 0, 0, 0};
    int a[6] = {0, 0, 0, 0, 0, 0};
    int cnt = 0;
    while (true)
    {
        cnt += 1;
        if (cnt % 100 == 0) printf("=====%d\n ", cnt);
        int m, n, k;
        float *A, *B, *C;

        get_inputs(&A, &B, &m, &n, &k);

        float *A_d, *B_d, *C_d;
        C = (float *)malloc(m * k * sizeof(float));
        hipMalloc(&A_d, m * n * sizeof(float));
        hipMalloc(&B_d, n * k * sizeof(float));
        hipMalloc(&C_d, m * k * sizeof(float));

        hipblasHandle_t handle;
        hipblasCreate(&handle);

        hipMemcpy(A_d, A, m * n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(B_d, B, n * k * sizeof(float), hipMemcpyHostToDevice);

        float alpha = 1.0f;
        float beta = 0.0f;

        for (int i = 0; i < 6; i++)
        {
            a[i] = set_val(m, n, k, state[i]);
            if (a[i] == 0)
                return 0;
        }

        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, a[0], a[1], a[2], &alpha, B_d, a[3], A_d, a[4], &beta, C_d, a[5]);
        // cublasSgemm(handle, CUBLAS_OP_N, CUBLAS_OP_N, m, m, n, &alpha, B_d, k, A_d, k, &beta, C_d, n);
        hipMemcpy(C, C_d, m * k * sizeof(float), hipMemcpyDeviceToHost);


       
        // verify(C, A, B, m, n, k);
       
        if (verify(C, A, B, m, n, k) == 1)
        {
            for (int i = 0; i < 6; i++)
                printf("%d ", a[i]);
            // return 0;
        }
        if (inc(state) == 0){
            printf("nop");
            return 0;
        }

        hipblasDestroy(handle);
        free(A);
        free(B);
        free(C);
        hipFree(A_d);
        hipFree(B_d);
        hipFree(C_d);

    }
    return 0;
}